
#include <hip/hip_runtime.h>
#include <cstddef>

#include <iostream>

#include <hipblas.h>
#include <hipsolver.h>

int main() {
    hipsolverHandle_t cusolverH;
    // cublasHandle_t cublasH;
    
    hipsolverDnCreate(&cusolverH);

    size_t size;
    scanf("%zu", &size);
    int
        *a_host = (int *) malloc(sizeof(int) * size),
        *b_host = (int *) malloc(sizeof(int) * size),
        *c_host = (int *) malloc(sizeof(int) * size);

    for (size_t i = 0; i < size; ++i)
    	scanf("%d", a_host + i);
    for (size_t i = 0; i < size; ++i)
    	scanf("%d", b_host + i);
    
    int *a_device, *b_device, *c_device;
    hipMalloc(&a_device, sizeof(int) * size);
    hipMalloc(&b_device, sizeof(int) * size);
    hipMalloc(&c_device, sizeof(int) * size);
    hipMemcpy(a_device, a_host, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, sizeof(int) * size, hipMemcpyHostToDevice);
    
    hipMemcpy(c_host, c_device, sizeof(int) * size, hipMemcpyDeviceToHost);
    for (size_t i = 0; i < size; ++i)
    	printf("%d ", c_host[i]);
    
    return 0;
}
