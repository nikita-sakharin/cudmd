#include <cstddef> // size_t

#include <stdexcept> // out_of_range

#include <hipblas.h>
#include <hipsolver.h> // cusolverDnXgesvd*
#include <hip/library_types.h> // HIP_C_64F, HIP_R_64F

#include <thrust/complex.h> // complex
#include <thrust/copy.h> // copy_n
#include <thrust/device_ptr.h> // device_ptr
#include <thrust/device_vector.h> // device_vector
#include <thrust/execution_policy.h> // device
#include <thrust/host_vector.h> // host_vector
#include <thrust/tuple.h> // make_tuple, tuple

#include <cudmd/cublas_helpers.h>
#include <cudmd/cudmd.h>
#include <cudmd/cusolverDn_helpers.h>
#include <cudmd/error_handling.h>
#include <cudmd/types.h>

using std::out_of_range;
using std::size_t;
using thrust::complex;
using thrust::copy_n;
using thrust::device;
using thrust::device_ptr;
using thrust::device_vector;
using thrust::host_vector;
using thrust::make_tuple;
using thrust::tuple;

__host__ tuple<
    device_vector<dbl>,
    device_vector<complex<dbl>>, device_vector<complex<dbl>>
> cudmd(
    const device_ptr<complex<dbl>> a_ptr,
    const size_t m, const size_t n, const size_t rank
) {
    const size_t k = n - 1;
    if (rank > k) [[unlikely]]
        throw out_of_range("cudmd: n must be greater than rank");

    cusolverDn_handle handle;

    device_vector<dbl> s_vector(k);
    device_vector<complex<dbl>> u_vector(m * k), vt_vector(k * k);
    size_t device_size, host_size;
    throw_if_error(cusolverDnXgesvd_bufferSize(
        handle.handle(), nullptr,
        'S', 'S', m, k,
        HIP_C_64F, a_ptr.get(), m,
        HIP_R_64F, s_vector.data().get(),
        HIP_C_64F, u_vector.data().get(), m,
        HIP_C_64F, vt_vector.data().get(), k,
        HIP_C_64F,
        &device_size, &host_size
    ), "cudmd: cusolverDnXgesvd_bufferSize");

    device_vector<char> device_workspace(device_size);
    host_vector<char> host_workspace(host_size);
    device_vector<int> device_info(1);
    throw_if_error(cusolverDnXgesvd(
        handle.handle(), nullptr,
        'S', 'S', m, k,
        HIP_C_64F, a_ptr.get(), m,
        HIP_R_64F, s_vector.data().get(),
        HIP_C_64F, u_vector.data().get(), m,
        HIP_C_64F, vt_vector.data().get(), k,
        HIP_C_64F,
        device_workspace.data().get(), device_size,
        host_workspace.data(), host_size,
        device_info.data().get()
    ), "cudmd: cusolverDnXgesvd");
    
    s_vector.resize(rank);
    u_vector.resize(m * rank);
    for (size_t i = 1; i < rank; ++i)
        copy_n(
            device,
            vt_vector.begin() + i * k, rank,
            vt_vector.begin() + i * rank
        );
    vt_vector.resize(rank * k);

    return make_tuple(s_vector, u_vector, vt_vector);
}
