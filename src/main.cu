#include "hip/hip_runtime.h"
#include <cassert>
#include <cstddef>

#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cudmd/error_handling.h>

using cublas_error = basic_error<hipblasStatus_t>;

__device__ inline int sum3(const int a, const int b, const int c) {
    return a + b + c;
}

__global__ __forceinline__ void hcd() {
    const int a = 5, b = 7, c = 11;
    const int d = sum3(a, b, c);
}

int main() {
    cublas_error
    	g(HIPBLAS_STATUS_SUCCESS, "4545"),
    	h(HIPBLAS_STATUS_SUCCESS, std::string("test")),
    	i(HIPBLAS_STATUS_SUCCESS),
    	j(HIPBLAS_STATUS_SUCCESS);
    std::cout << g.what() << '\n' << i.code();
    hcd<<<1, 1>>>();

    return 0;
}
