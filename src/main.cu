#include <cassert>
#include <cstddef>

#include <iostream>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <cudmd/error_handling.h>

using cublas_error = basic_error<hipblasStatus_t>;

int main() {
    cublas_error
    	g(HIPBLAS_STATUS_SUCCESS, "4545"),
    	h(HIPBLAS_STATUS_SUCCESS, std::string("test")),
    	i(HIPBLAS_STATUS_SUCCESS),
    	j(HIPBLAS_STATUS_SUCCESS);
    std::cout << g.what() << '\n' << i.code();

    return 0;
}
